


#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

#define N 512


__global__ void minElement (int *a,int *o)
{
    int of = N/2;

    int tid = threadIdx.x;

    for(of;of>0;of = of/2)
    {
        if(tid < of)
        {
            if(a[tid] > a[tid+of])
            {
                a[tid] = a[tid+of];
            }
        }

    }

    o[0] = a[0];
}

int main()
{
    int *h_a,*d_a,*o_a,*oh_a;
    int size = N*sizeof(int);
    h_a = (int *)malloc(size);
    oh_a = (int *)malloc(size);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&o_a,size);

    for(int i = 1; i <= N;i++)
    {
        h_a[i-1] = i;
    }

    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);

    minElement<<<1,N/2>>>(d_a,o_a);

    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);

    cout<<"Min Element is - "<<oh_a[0];

    hipFree(d_a);
    free(h_a);
}

