
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

#define N 512


__global__ void ArithmeticMean (int *a,int *o)
{
    int of = N/2;

    int tid = threadIdx.x;

    for(of;of>0;of = of/2)
    {
        if(tid < of)
        {
            a[tid]+=a[tid+of];
        }
    }

    o[0] = a[0];
}

int main()
{
    int *h_a,*d_a,*o_a,*oh_a;
    int size = N*sizeof(int);
    h_a = (int *)malloc(size);
    oh_a = (int *)malloc(size);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&o_a,size);

    for(int i = 1; i <= N;i++)
    {
        h_a[i-1] = i;
    }


    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);

    ArithmeticMean<<<1,N/2>>>(d_a,o_a);

    hipMemcpy(h_a,d_a,size,hipMemcpyDeviceToHost);
    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);
    
    float AM =(float) oh_a[0]/N;
    cout<<"AM is "<<AM;

    hipFree(d_a);
    free(h_a);
}

