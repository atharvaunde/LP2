
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

#define N 512


__global__ void Sum (int *a,int *o)
{
    int of = N/2;

    int tid = threadIdx.x;

    for(of;of>0;of = of/2)
    {
        if(tid < of)
        {
            a[tid]+=a[tid+of];
        }

    }

    o[0] = a[0];
}

__global__ void standardDeviation(int *a,int avg)
{
  int tid = threadIdx.x;
  if(tid<N)
  {
    a[tid] -= avg;
    a[tid] = a[tid]*a[tid];
  }
}

int main()
{
    int *h_a,*d_a,*o_a,*oh_a,*d_a1;
    int size = N*sizeof(int);
    h_a = (int *)malloc(size);
    oh_a = (int *)malloc(size);

    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&o_a,size);
    //new
    hipMalloc((void**)&d_a1,size);

    for(int i = 1; i <= N;i++)
    {
        h_a[i-1] = i;
    }


    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_a1,h_a,size,hipMemcpyHostToDevice);

    Sum<<<1,N/2>>>(d_a,o_a);

    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);

    int arithmetcMean = oh_a[0]/N;

    standardDeviation<<<1,N>>>(d_a1,arithmetcMean);

    Sum<<<1,N/2>>>(d_a1,o_a);

    hipMemcpy(oh_a,o_a,size,hipMemcpyDeviceToHost);

    int tmp = oh_a[0]/N;

    cout<<"Standard Deviation is - "<<sqrt(tmp)<<endl;

    hipFree(d_a);
    free(h_a);
    hipFree(o_a);
    free(oh_a);
    hipFree(d_a1);
    
}

