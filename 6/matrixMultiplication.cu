
#include <hip/hip_runtime.h>
#include<iostream>
#include <math.h>
using namespace std;

#define N 10

__global__ void fMatrixMultiplication (int *A,int *B,int *C)
{
  int ROW = blockIdx.y*blockDim.y+threadIdx.y;
  int COL = blockIdx.x*blockDim.x+threadIdx.x;

  int tmpSum = 0;

  if (ROW < N && COL < N) {
      // each thread computes one element of the block sub-matrix
      for (int i = 0; i < N; i++) {
          tmpSum += A[ROW * N + i] * B[i * N + COL];
      }
    C[ROW * N + COL] = tmpSum;
    }
  
}

int main()
{
    int *h_A,*h_B,*h_C;
    int cnt=1;
    int size = N*N*sizeof(int);
    h_A = (int*) malloc(size);
    h_B = (int*) malloc(size);
    h_C = (int*) malloc(size);

    // Initialize matrices on the host
    for (int i=1; i<=N*N; i++){
            if(cnt==N+1)
            	cnt = 1;
            h_A[i-1] = cnt;
            h_B[i-1] = cnt;
            cnt+=1;
    }

    cout<<"Matrix A"<<endl;
    for (int i=0; i<N*N; i++){
	     cout<<h_A[i]<<" ";
	     if(i%N==N-1)
		cout<<endl;
    }
    cout<<endl;
    cout<<"Matrix B"<<endl;
    for (int i=0; i<N*N; i++){
	     cout<<h_B[i]<<" ";
	     if(i%N==N-1)
		cout<<endl;
    }
    cout<<endl;

    int *d_A,*d_B,*d_C;
    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);

    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
     
    fMatrixMultiplication<<<blocksPerGrid,threadsPerBlock>>>(d_A,d_B,d_C);

    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
    
    cout<<"Matrix C = A*B"<<endl;
        
   for (int i=0; i<N*N; i++){
	     cout<<h_C[i]<<" ";
	     if(i%N==N-1)
		cout<<endl;
    }
    hipFree(d_A);
    free(h_A);
    hipFree(d_B);
    free(h_B);
    hipFree(d_C);
    free(h_C);
}


/*
1234
1234
1234
1234
*/

