#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define n 512

__global__ void bmk_add(int *a, int *b, int *result)
{
	int i = threadIdx.x;
	result[i] = a[i] + b[i];
}

int main()
{
	int num_blocks = 1, num_threads = n;

	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;

	int size = n * sizeof(int);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	hipMalloc((void**)&dev_a,size);
	hipMalloc((void**)&dev_b,size);
	hipMalloc((void**)&dev_c,size);

	for(int i = 0;i<n;i++)
	{
		//a[i] = rand()%1024;
		//b[i] = rand()%1024;
		a[i] = i;
		b[i] = i;
	}

	hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size,hipMemcpyHostToDevice);

	bmk_add <<<num_blocks, num_threads>>>(dev_a,dev_b,dev_c);

	hipMemcpy(c,dev_c,size,hipMemcpyDeviceToHost);

	for(int i = 0;i<n;i++)
		printf("%d  ",c[i]);
    
    printf("\n");
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
